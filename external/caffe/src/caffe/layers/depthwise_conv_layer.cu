#include "hip/hip_runtime.h"
#include <vector>
#include <algorithm>
#include <cfloat>
#include "caffe/layers/depthwise_conv_layer.hpp"
#include "caffe/util/math_functions.hpp"


/*
 * The depthwise layer for mobilenet.   only for stride 1
 */

namespace caffe {

template <typename Dtype>
__global__ void ConvForward(const int nthreads,
		const Dtype* const bottom_data, const int num, const int channels,
		const int height, const int width,const int conved_height,
		const int conved_width,const int kernel_h, const int kernel_w,
		const int stride_h, const int stride_w, const int pad_h, const int pad_w,
		Dtype* const top_data,const Dtype* const weight,const Dtype* const bias,const bool bias_term_) {
	CUDA_KERNEL_LOOP(index, nthreads) {

		const int pw = index % conved_width;
		const int ph = (index / conved_width) % conved_height;
		const int c = (index / conved_width / conved_height) % channels;
		const int n = index / conved_width / conved_height / channels;
		int hstart = ph * stride_h - pad_h;
		int wstart = pw * stride_w - pad_w;
		int hend = min(hstart + kernel_h, height + pad_h);
		int wend = min(wstart + kernel_w, width + pad_w);
//		const int pool_size = (hend - hstart) * (wend - wstart);
		hstart = max(hstart, 0);
		wstart = max(wstart, 0);
		hend = min(hend, height);
		wend = min(wend, width);
		Dtype aveval = 0;
		const Dtype* const bottom_slice =
		bottom_data + (n * channels + c) * height * width;
		const Dtype* const weight_slice =
		weight + c * kernel_h * kernel_w;
//		if (index==1) {
//			printf("pw%d ph%d c%d n%d \n",pw,ph,c,n);
//			printf("hstart%d wstart%d hend%d wend%d \n",hstart,wstart,hend,wend);
//		}

		int khstart=hend<kernel_h?kernel_h-hend:0;
		int kwstart=wend<kernel_w?kernel_w-wend:0;
		for (int h = hstart; h < hend; ++h) {
			for (int w = wstart; w < wend; ++w) {

				aveval += bottom_slice[h * width + w]*weight_slice[(khstart+h-hstart) * kernel_w + (kwstart+w-wstart)];
//				if (index==1) {
//					printf("pos:h%d w%d\n",h,w);
//					printf("cal:bottom%f weight%f\n",bottom_slice[h * width + w],weight_slice[(h-hstart) * kernel_w + (w-wstart)]);
//				}
			}
		}
		if(bias_term_) {
			aveval+=bias[c];
		}
		top_data[index] = aveval;
	}
}

template<typename Dtype>
void DepthwiseConvolutionLayer<Dtype>::Forward_gpu(
		const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
//	std::cout << "fp" << std::endl;
	const Dtype* weight = this->blobs_[0]->gpu_data();
	int* kernel_shape_data = this->kernel_shape_.mutable_cpu_data();
	int* stride_data = this->stride_.mutable_cpu_data();
	int* pad_data = this->pad_.mutable_cpu_data();

	for (int i = 0; i < bottom.size(); ++i) {
		const Dtype* bottom_data = bottom[i]->gpu_data();
		Dtype* top_data = top[i]->mutable_gpu_data();
		const int count = top[i]->count();
		vector<int> shape_ = bottom[i]->shape();
		const int channels_ = shape_[1];
		const int height_ = shape_[2];
		const int width_ = shape_[3];

		const int kernel_h_ = kernel_shape_data[0];
		const int kernel_w_ = kernel_shape_data[1];
		const int stride_h_ = stride_data[0];
		const int stride_w_ = stride_data[1];
		const int pad_h_ = pad_data[0];
		const int pad_w_ = pad_data[1];

		const int conved_height = this->output_shape_[0];
		const int conved_weight = this->output_shape_[1];

		const bool bias_term_ = this->bias_term_;

		if (bias_term_) {
			const Dtype* const bias = this->blobs_[1]->gpu_data();
			ConvForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
					count, bottom_data, bottom[i]->num(), channels_,
					height_, width_,conved_height,conved_weight,kernel_h_,
					kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data,weight,bias,bias_term_);
		} else {
			ConvForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
					count, bottom_data, bottom[i]->num(), channels_,
					height_, width_,conved_height,conved_weight,kernel_h_,
					kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data,weight,0,bias_term_);
		}
	}
}

template <typename Dtype>
__global__ void ConvBackward(const int nthreads,
const Dtype* const top_diff,
const int num, const int channels, const int height,
const int width, const int conved_height, const int conved_width,
const int kernel_h, const int kernel_w, const int stride_h,
const int stride_w, const int pad_h, const int pad_w,
Dtype* const bottom_diff,
const Dtype* const weight) {

	CUDA_KERNEL_LOOP(index, nthreads) {
		const int w = index % width + pad_w;
		const int h = (index / width) % height + pad_h;
		const int c = (index / width / height) % channels;
		const int n = index / width / height / channels;
		
		const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
		const int phend = min(h / stride_h + 1, conved_height);
		const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
		const int pwend = min(w / stride_w + 1, conved_width);
		
		const int khstart=(h >= kernel_h) ? ((h-kernel_h)%stride_h)+(kernel_h-stride_h): h;
		const int kwstart=(w >= kernel_w) ? ((w-kernel_w)%stride_w)+(kernel_w-stride_w) : w;
		
		Dtype gradient = 0;
		const Dtype* const top_diff_slice =
		top_diff + (n * channels + c) * conved_height * conved_width;
		
		const Dtype* const weight_slice =weight + c * kernel_h * kernel_w;
		
//		if (index==2) {
//			printf("w%d h%d c%d n%d \n",w,h,c,n);
//			printf("phstart%d phend%d pwstart%d pwend%d \n",phstart,phend,pwstart,pwend);
//		}
		
		for (int ph = phstart; ph < phend; ++ph) {
			for (int pw = pwstart; pw < pwend; ++pw) {
				int kh=khstart-(ph-phstart)*stride_h;
				int kw=kwstart-(pw-pwstart)*stride_w;
				gradient += top_diff_slice[ph * conved_width + pw] *weight_slice[kh*kernel_w+kw];
				
//						if (index==2) {
//							printf("pos:ph%d pw%d kh%d kw%d\n",ph,pw,kh,kw);
//							printf("cal:top_diff%f weight%f\n",top_diff_slice[ph * conved_width + pw],weight_slice[kh*kernel_w+kw]);
//				//			printf("cal:top_diff%f weight%f\n",top_diff_slice[ph * conved_width + pw],weight_slice[kh*kernel_w+kw]);
//						}
			}
		}
		bottom_diff[index] = gradient;
	}
}

__device__ float atomicAddme(float* address, float val)
{
    return atomicAdd(address,val);
}

__device__ double atomicAddme(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + 
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}



#define DIVIDE_CEIL(a,b) a/b+((a/b*b)<a)


template <typename Dtype>
__global__ void ConvBackwardWeight(const int nthreads,
const Dtype* const top_diff,
const int num, const int channels, const int height,
const int width, const int conved_height, const int conved_width,
const int kernel_h, const int kernel_w, const int stride_h,
const int stride_w, const int pad_h, const int pad_w,
Dtype* const weight_diff,
const Dtype* const bottom_data) {

	CUDA_KERNEL_LOOP(index, nthreads) {
		const int kw=index % kernel_w;
		const int kh= (index /kernel_w)%kernel_h;
		const int c=index /kernel_w/kernel_h;
		
//		if (index==5) {
//			printf("kh%d kw%d kc%d\n",kh,kw,c);
//		}
		Dtype gradient = 0;
		for( int n=0;n<num;n++) {
			
			const Dtype* const top_diff_slice = top_diff + (n * channels + c) * conved_height * conved_width;
			const Dtype* const bottom_data_slice = bottom_data + (n * channels + c) * height * width;
		
			
			const int phstart=max(DIVIDE_CEIL((pad_h-kh),stride_h),0);
			const int phend=min(DIVIDE_CEIL((height+pad_h-kh),stride_h),conved_height);
		
			const int pwstart=max(DIVIDE_CEIL((pad_w-kw),stride_w),0);
			
			const int pwend=min(DIVIDE_CEIL((width+pad_w-kw),stride_w),conved_width);
//			if (index==5) {
//				printf("phstart%d phend%d pwstart%d pwend%d \n",phstart,phend,pwstart,pwend);
//			}
//			
			for(int ph=phstart;ph<phend;ph++){
				for (int pw=pwstart;pw<pwend;pw++){
					const int h=ph*stride_h+kh-pad_h;
					const int w=pw*stride_w+kw-pad_w;
					gradient+=top_diff_slice[ph * conved_width + pw]*bottom_data_slice[h*width+w];
//					if (index==5) {
//						printf("n%d h%d w%d ph%d pw%d topdiff%f bottomdata%f\n",n,h,w,ph,pw,top_diff_slice[ph * conved_width + pw],bottom_data_slice[h*width+w]);
//			//			printf("phstart%d phend%d pwstart%d pwend%d \n",phstart,phend,pwstart,pwend);
//					}
				}
			}
		}
		weight_diff[c * kernel_h * kernel_w+kh*kernel_w+kw]+=gradient;
	}
}

template <typename Dtype>
__global__ void ConvBackwardBias(const int nthreads,
const Dtype* const top_diff,
const int num, const int channels, const int height,
const int width, const int conved_height, const int conved_width,
const int kernel_h, const int kernel_w, const int stride_h,
const int stride_w, const int pad_h, const int pad_w,
Dtype* const bias_diff) {
	CUDA_KERNEL_LOOP(index, nthreads) {
		const int c = index;
		Dtype gradient=0;
		for( int n=0;n<num;n++) {
			const Dtype* const top_diff_slice =
			top_diff + (n * channels + c) * conved_height * conved_width;
			for(int ph=0;ph<conved_height;ph++) {
				for (int pw=0;pw<conved_width;pw++) {
					gradient+=top_diff_slice[ph * conved_width + pw];
				}
			}
		}
		bias_diff[c]+=gradient;
	}
}
template<typename Dtype>
void DepthwiseConvolutionLayer<Dtype>::Backward_gpu(
const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
const vector<Blob<Dtype>*>& bottom) {


	int* kernel_shape_data = this->kernel_shape_.mutable_cpu_data();
	int* stride_data = this->stride_.mutable_cpu_data();
	int* pad_data = this->pad_.mutable_cpu_data();

	const Dtype* weight = this->blobs_[0]->gpu_data();
	Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();

	const bool bias_term_ = this->bias_term_;
	Dtype* bias_diff = bias_term_ ? this->blobs_[1]->mutable_gpu_diff() : 0;
	const bool bias_propagate_down_ = this->param_propagate_down_[1];
	const bool weight_propagate_down_ = this->param_propagate_down_[0];


	const int kernel_h_ = kernel_shape_data[0];
	const int kernel_w_ = kernel_shape_data[1];
	const int stride_h_ = stride_data[0];
	const int stride_w_ = stride_data[1];
	const int pad_h_ = pad_data[0];
	const int pad_w_ = pad_data[1];

	const int conved_height = this->output_shape_[0];
	const int conved_weight = this->output_shape_[1];

//	CHECK_EQ(stride_h_, 1)
//	        << "The backward of the net whose stride is bigger than 1 is not implemented now. ";
//	CHECK_EQ(stride_w_, 1)
//	        << "The backward of the net whose stride is bigger than 1 is not implemented now. ";


	for (int i = 0; i < top.size(); ++i) {

		const Dtype* top_diff = top[i]->gpu_diff();
		const Dtype* bottom_data = bottom[i]->gpu_data();
		Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();

		vector<int> shape_ = bottom[i]->shape();
		const int channels_ = shape_[1];
		const int height_ = shape_[2];
		const int width_ = shape_[3];

		// Bias gradient, if necessary.
		if (bias_term_ && bias_propagate_down_) {
			const int count_bias = channels_;
			ConvBackwardBias<Dtype><<<CAFFE_GET_BLOCKS(count_bias), CAFFE_CUDA_NUM_THREADS>>>(
				count_bias, top_diff, bottom[i]->num(), channels_,
				height_, width_,conved_height,conved_weight,kernel_h_,
				kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
				bias_diff);
		}
		// gradient w.r.t. weight. Note that we will accumulate diffs.
		if (weight_propagate_down_) {
			const int count_weight = channels_ * kernel_h_ * kernel_w_;
			ConvBackwardWeight<Dtype><<<CAFFE_GET_BLOCKS(count_weight), CAFFE_CUDA_NUM_THREADS>>>(
					count_weight, top_diff, bottom[i]->num(), channels_,
				height_, width_,conved_height,conved_weight,kernel_h_,
				kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
				weight_diff,
				bottom_data);
		}
		// gradient w.r.t. bottom data, if necessary.
		if (propagate_down[i]) {
			const int count_bottom=bottom[i]->count();
			ConvBackward<Dtype><<<CAFFE_GET_BLOCKS(count_bottom), CAFFE_CUDA_NUM_THREADS>>>(
				count_bottom, top_diff, bottom[i]->num(), channels_,
				height_, width_,conved_height,conved_weight,kernel_h_,
				kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, 
				bottom_diff,
				weight);
		}
	}

}

INSTANTIATE_LAYER_GPU_FUNCS (DepthwiseConvolutionLayer);

}  // namespace caffe
