#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/softmax_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SoftmaxLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, const Dtype* weight, Dtype* loss,
          const int num, const int dim, const int spatial_dim,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index] = 0;
      counts[index] = 0;
    } else {
    
      // gb added
      const Dtype weight_value = (weight != NULL) ? static_cast<Dtype>(weight[n * spatial_dim + s]) : 1;
      
      loss[index] = - weight_value * log(max(prob_data[n * dim + label_value * spatial_dim + s],
                      Dtype(FLT_MIN)));
                      
      counts[index] = weight_value;
    }
  }
}

template <typename Dtype>
void SoftmaxWithLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const int dim = prob_.count() / outer_num_;
  const int nthreads = outer_num_ * inner_num_;
  
  // gb added
  bool has_weight = bottom.size() >= 3;
  const Dtype* weight = NULL;
  if (has_weight)
      weight = bottom[2]->gpu_data();
  
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  // Similarly, this memory is never used elsewhere, and thus we can use it
  // to avoid having to allocate additional GPU memory.
  Dtype* counts = prob_.mutable_gpu_diff();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SoftmaxLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, label, weight, loss_data,
      outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);
  Dtype loss;
  caffe_gpu_asum(nthreads, loss_data, &loss);
  Dtype valid_count = -1;
  // Only launch another CUDA kernel if we actually need the count of valid
  // outputs.
  //if (normalization_ == LossParameter_NormalizationMode_VALID &&
  //    has_ignore_label_) {
  //  caffe_gpu_asum(nthreads, counts, &valid_count);
  //}
  
  //top[0]->mutable_cpu_data()[0] = loss / get_normalizer(normalization_,
  //                                                      valid_count);
  
  // gb added (disable above, add below)
  caffe_gpu_asum(nthreads, counts, &valid_count);
  loss /= valid_count;

  top[0]->mutable_cpu_data()[0] = loss;
  if (top.size() == 2) {
    top[1]->ShareData(prob_);
  }
}

template <typename Dtype>
__global__ void SoftmaxLossBackwardGPU(const int nthreads, const Dtype* top,
          const Dtype* label, const Dtype *weight, Dtype* bottom_diff, const int num, const int dim,
          const int spatial_dim, const bool has_ignore_label_,
          const int ignore_label_, Dtype* counts) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);

    if (has_ignore_label_ && label_value == ignore_label_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
    } else {
      bottom_diff[n * dim + label_value * spatial_dim + s] -= 1;
      if (weight != NULL){
          const Dtype weight_value = static_cast<Dtype>(weight[n * spatial_dim + s]);
		  for (int k = 0; k < channels; ++k)
              bottom_diff[n * dim + k * spatial_dim + s] *= weight_value;

          counts[index] = weight_value;
	  }
	  else
          counts[index] = 1;
    }
  }
}

template <typename Dtype>
void SoftmaxWithLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* top_data = top[0]->gpu_data();
    caffe_gpu_memcpy(prob_.count() * sizeof(Dtype), prob_data, bottom_diff);
    const Dtype* label = bottom[1]->gpu_data();
    const int dim = prob_.count() / outer_num_;
    const int nthreads = outer_num_ * inner_num_;
    
    // gb added
    bool has_weight = bottom.size() >= 3;
	const Dtype* weight = NULL;
	if (has_weight)
		weight = bottom[2]->gpu_data();

    // Since this memory is never used for anything else,
    // we use to to avoid allocating new GPU memory.
    Dtype* counts = prob_.mutable_gpu_diff();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SoftmaxLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, top_data, label, weight, bottom_diff,
        outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);

    Dtype valid_count = -1;
    // Only launch another CUDA kernel if we actually need the count of valid
    // outputs.
    //if (normalization_ == LossParameter_NormalizationMode_VALID &&
    //    has_ignore_label_) {
    //  caffe_gpu_asum(nthreads, counts, &valid_count);
    //}
    //const Dtype loss_weight = top[0]->cpu_diff()[0] /
    //                          get_normalizer(normalization_, valid_count);
    
    caffe_gpu_asum(nthreads, counts, &valid_count);
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    caffe_gpu_scal(prob_.count(), valid_count == Dtype(0.0) ? Dtype(0.0) : (loss_weight / valid_count), bottom_diff);
    
    //caffe_gpu_scal(prob_.count(), loss_weight , bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxWithLossLayer);

}  // namespace caffe
