#include "hip/hip_runtime.h"
/*
* Example of how to use the mxGPUArray API in a MEX file.  This example shows
* how to write a MEX function that takes a gpuArray input and returns a
* gpuArray output, e.g. B=mexFunction(A).
*
* Copyright 2012 The MathWorks, Inc.
*/

#include "mex.h"
#include <vector>
#include <iostream>

#define DIVUP(m,n)		((m)/(n)+((m)%(n)>0))
int const threadsPerBlock = (sizeof(unsigned long long) * 8);

/*
* Device code
*/
__device__ inline float devCheck(float const * const a, float const * const b, float const nms_overlap_thres, float const inside_thres)
{
	float left = max(a[0], b[0]), right = min(a[2], b[2]);
	float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
	float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
	float interS = width * height;
	float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
	float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
	// || interS/Sa > inside_thres || interS/Sb > inside_thres
	return interS / (Sa + Sb - interS) > nms_overlap_thres;
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thres, const float inside_thres, const float *dev_boxes, unsigned long long *dev_mask)
{
	const int row_start = blockIdx.y, col_start = blockIdx.x;
	const int row_size = min(n_boxes - row_start * threadsPerBlock, threadsPerBlock), col_size = min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

	//if (row_start > col_start) return;

	__shared__ float block_boxes[threadsPerBlock * 5];
	if (threadIdx.x < col_size)
	{
		block_boxes[threadIdx.x * 5 + 0] = dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
		block_boxes[threadIdx.x * 5 + 1] = dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
		block_boxes[threadIdx.x * 5 + 2] = dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
		block_boxes[threadIdx.x * 5 + 3] = dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
		block_boxes[threadIdx.x * 5 + 4] = dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
	}
	__syncthreads();

	if (threadIdx.x < row_size)
	{
		const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
		const float *cur_box = dev_boxes + cur_box_idx * 5;
		int i = 0;
		unsigned long long t = 0;
		int start = 0;
		if (row_start == col_start) start = threadIdx.x + 1;
		for (i = start; i < col_size; i++)
		{
			if (devCheck(cur_box, block_boxes + i * 5, nms_overlap_thres, inside_thres)){
				t |= 1ULL << i;
			}
		}
		const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
		dev_mask[cur_box_idx * col_blocks + col_start] = t;
	}
}

/*
* Host code
*/
void mexFunction(int nlhs, mxArray *plhs[],
	int nrhs, const mxArray *prhs[])
{
	
	/* Declare all variables.*/
	mxArray const *boxes, *ov_thres, *ov_inside;
	float *boxes_host = NULL;
	float *boxes_dev = NULL;
	unsigned long long *mask_dev = NULL;

	/* Throw an error if the input is not a array. */
	if (nrhs != 3) {
		mexErrMsgTxt("nms_gpu_mex::need 2 inputs");
	}

	boxes = prhs[0];
	if (mxGetClassID(boxes) != mxSINGLE_CLASS) {
		mexErrMsgTxt("nms_gpu_mex::input boxes must be single");
	}

	ov_thres = prhs[1];
	if (mxGetClassID(ov_thres) != mxDOUBLE_CLASS) {
		mexErrMsgTxt("nms_gpu_mex::thres must be double");
	}
	
	ov_inside = prhs[2];
	if (mxGetClassID(ov_inside) != mxDOUBLE_CLASS) {
		mexErrMsgTxt("nms_gpu_mex::inside thres must be double");
	}

	float nms_overlap_thres = (float)mxGetScalar(ov_thres);
	float inside_thres = (float)mxGetScalar(ov_inside);

	int boxes_dim = mxGetM(boxes);
	int boxes_num = mxGetN(boxes);
	if (boxes_dim != 5)
	{
		mexErrMsgTxt("nms_gpu_mex::input boxes's row must be 5");
	}

	boxes_host = (float *)(mxGetPr(boxes));
	const int col_blocks = DIVUP(boxes_num, threadsPerBlock);

	hipMalloc(&boxes_dev, mxGetNumberOfElements(boxes) * sizeof(float));
	hipMemcpy(boxes_dev, boxes_host, mxGetNumberOfElements(boxes) * sizeof(float), hipMemcpyHostToDevice);

	/* Create a GPUArray to hold the result and get its underlying pointer. */
	hipMalloc(&mask_dev, boxes_num * col_blocks * sizeof(unsigned long long));
	

	/*
	* Call the kernel using the CUDA runtime API. We are using a 1-d grid here,
	* and it would be possible for the number of elements to be too large for
	* the grid. For this example we are not guarding against this possibility.
	*/

	dim3 blocks(DIVUP(boxes_num, threadsPerBlock), DIVUP(boxes_num, threadsPerBlock));
	dim3 threads(threadsPerBlock);
	nms_kernel << <blocks, threads >> >(boxes_num, nms_overlap_thres, inside_thres, boxes_dev, mask_dev);

	std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
	hipMemcpy(&mask_host[0], mask_dev, sizeof(unsigned long long) * boxes_num * col_blocks, hipMemcpyDeviceToHost);

	std::vector<unsigned long long> remv(col_blocks);
	memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

	std::vector<int> keep;
	keep.reserve(boxes_num);
	for (int i = 0; i < boxes_num; i++)
	{
		int nblock = i / threadsPerBlock;
		int inblock = i % threadsPerBlock;

		if (!(remv[nblock] & (1ULL << inblock)))
		{
			keep.push_back(i + 1);  // to matlab's index

			unsigned long long *p = &mask_host[0] + i * col_blocks;
			for (int j = nblock; j < col_blocks; j++)
			{
				remv[j] |= p[j];
			}
		}
	}

	/* Wrap the result up as a MATLAB cpuArray for return. */
	mwSize dims[4] = { (int)keep.size(), 1, 1, 1 };
	plhs[0] = mxCreateNumericArray(4, dims, mxINT32_CLASS, mxREAL);

	int *output = (int *)(mxGetPr(plhs[0]));
	memcpy(output, &keep[0], (int)keep.size() * sizeof(int));


	hipFree(boxes_dev);
	hipFree(mask_dev);
}
